#include "hip/hip_runtime.h"
#include "vertex_ops.cuh"

__global__
void unproject_kernel(unsigned char *depth, double* d_3d_points, int rows, int cols, double cx, double cy, double fx, double fy, double fx_inv, double fy_inv)
{

	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int idy = threadIdx.y + (blockIdx.y * blockDim.y);
	int id;
	if ((idx < rows ) && (idy < cols))
	{
		// add depth info
		id = idx * idy;
		d_3d_points[id] = (idx-cx)*fx_inv;
		d_3d_points[id+(rows*cols)] = (idy-cy)*fy_inv;
		d_3d_points[id+2*(rows*cols)] = 1;
		// printf("--%f--\n",(cx));
	}
}


void unproject(cv::Mat img, GSLAM::CameraPinhole cam)
{

	// uchar4 **ddepth, **input_image;
	// unsigned char *hdepth;
	unsigned char *d_depth_image;
	double *d_3d_points, *h_3d_points;
	int rows, cols;
	rows = img.rows;
	cols = img.cols;
	size_t totalpixels = rows*cols;
	const dim3 dimGrid((int)ceil((cols)/16), (int)ceil((rows)/16));
	const dim3 dimBlock(16, 16);
	int size_[] = { rows,cols,3 };
	// *input_image = (uchar4 *)img.ptr<uchar4 *>(0);
	// hipMalloc(ddepth, sizeof(uchar4) * totalpixels * CHANNELS);
	// hipMemcpy(*ddepth, *input_image, sizeof(uchar4) * totalpixels * CHANNELS, hipMemcpyHostToDevice);

	h_3d_points = (double*)malloc(sizeof(double) * totalpixels * 3);
	unsigned char* depth_image = (unsigned char*)img.data;
	hipMalloc((void **)&d_depth_image, sizeof(unsigned char) * totalpixels );
	hipMalloc((void **)&d_3d_points, sizeof(double) * totalpixels * 3 );
	hipMemcpy(d_depth_image, depth_image, sizeof(unsigned char) * totalpixels , hipMemcpyHostToDevice);
	unproject_kernel<<<dimGrid,dimBlock>>>(d_depth_image, d_3d_points, rows, cols, cam.cx, cam.cy, cam.fx, cam.fy, cam.fx_inv, cam.fy_inv);
	hipMemcpy(h_3d_points, d_3d_points, sizeof(double) * totalpixels * 3, hipMemcpyDeviceToHost);
	
	std::cout<<*(h_3d_points+sizeof(double)+2*(rows*cols));
	hipFree(d_depth_image);
}