#include "hip/hip_runtime.h"
#include "vertex_ops.cuh"


texture<float, 2, hipReadModeElementType> texRef; 

__global__
void unproject_kernel(unsigned char *depth, double* d_3d_points, int rows, int cols, double cx, double cy, double fx, double fy, double fx_inv, double fy_inv)
{

	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int idy = threadIdx.y + (blockIdx.y * blockDim.y);
	int id;
	if ((idx < rows ) && (idy < cols))
	{
		// add depth info
		id = idx * idy;
		double z = (double)depth[id];
		d_3d_points[id] = (idx-cx)*fx_inv*z;
		d_3d_points[id+(rows*cols)] = (idy-cy)*fy_inv*z;
		d_3d_points[id+2*(rows*cols)] = z;
		// printf("--%f--\n",(cx));
	}
}


void unproject(cv::Mat img, GSLAM::CameraPinhole cam)
{

	// uchar4 **ddepth, **input_image;
	// unsigned char *hdepth;
	unsigned char *d_depth_image;
	double *d_3d_points, *h_3d_points;
	int rows, cols;
	rows = img.rows;
	cols = img.cols;
	size_t totalpixels = rows*cols;
	const dim3 dimGrid((int)ceil((cols)/16), (int)ceil((rows)/16));
	const dim3 dimBlock(16, 16);
	std::cout<<"t";
	// *input_image = (uchar4 *)img.ptr<uchar4 *>(0);
	// hipMalloc(ddepth, sizeof(uchar4) * totalpixels * CHANNELS);
	// hipMemcpy(*ddepth, *input_image, sizeof(uchar4) * totalpixels * CHANNELS, hipMemcpyHostToDevice);

	// h_3d_points = (double*)malloc(sizeof(double) * totalpixels * 3);
	// unsigned char* depth_image = (unsigned char*)img.data;
	// hipMalloc((void **)&d_depth_image, sizeof(unsigned char) * totalpixels );
	// hipMalloc((void **)&d_3d_points, sizeof(double) * totalpixels * 3 );
	// hipMemcpy(d_depth_image, depth_image, sizeof(unsigned char) * totalpixels , hipMemcpyHostToDevice);
	// unproject_kernel<<<dimGrid,dimBlock>>>(d_depth_image, d_3d_points, rows, cols, cam.cx, cam.cy, cam.fx, cam.fy, cam.fx_inv, cam.fy_inv);
	// hipMemcpy(h_3d_points, d_3d_points, sizeof(double) * totalpixels * 3, hipMemcpyDeviceToHost);
	
	// std::cout<<*(h_3d_points+sizeof(double)+(rows*cols));
	// hipFree(d_depth_image);

	texture<double, hipTextureType2D,  hipReadModeElementType> t;
	hipArray* cuArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,
                                             hipChannelFormatKindFloat );
    hipMallocArray(&cuArray, &channelDesc, cols, rows);
    hipMemcpyToArray(cuArray, 0, 0, img.data, sizeof(double)*totalpixels, hipMemcpyHostToDevice);

      
}

void rgb_texture_test(cv::Mat img)
{
	int width = img.cols;
	int height = img.rows;
	int size = width * height  * sizeof(float);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
	hipArray* cuArray;
	cudaSafeCall(hipMallocArray(&cuArray, &channelDesc, width, height));
	cudaSafeCall(hipMemcpyToArray(cuArray, 0, 0, img.data, size, hipMemcpyHostToDevice));

	texRef.addressMode[0] = hipAddressModeWrap;
	texRef.addressMode[1] = hipAddressModeWrap;
	texRef.filterMode = hipFilterModeLinear;
	texRef.normalized = true;

	hipBindTextureToArray(texRef, cuArray, channelDesc);

    cudaCheckError();
    std::cout<<"working\n";
    hipFreeArray(cuArray);

}