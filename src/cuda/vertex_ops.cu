#include "hip/hip_runtime.h"
#include "vertex_ops.cuh"



__global__
void unproject_kernel(unsigned char *depth, double* d_3d_points, int rows, int cols, double cx, double cy, double fx, double fy, double fx_inv, double fy_inv)
{

	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int idy = threadIdx.y + (blockIdx.y * blockDim.y);
	int id;
	if ((idx < rows ) && (idy < cols))
	{
		// add depth info
		id = idx * idy;
		double z = (double)depth[id];
		d_3d_points[id] = (idx-cx)*fx_inv*z;
		d_3d_points[id+(rows*cols)] = (idy-cy)*fy_inv*z;
		d_3d_points[id+2*(rows*cols)] = z;
		// printf("--%f--\n",(cx));
	}
}


void unproject(cv::Mat img, GSLAM::CameraPinhole cam)
{

	// uchar4 **ddepth, **input_image;
	// unsigned char *hdepth;
	unsigned char *d_depth_image;
	double *d_3d_points, *h_3d_points;
	int rows, cols;
	rows = img.rows;
	cols = img.cols;
	size_t totalpixels = rows*cols;
	const dim3 dimGrid((int)ceil((cols)/16), (int)ceil((rows)/16));
	const dim3 dimBlock(16, 16);
	std::cout<<"t";
	// *input_image = (uchar4 *)img.ptr<uchar4 *>(0);
	// hipMalloc(ddepth, sizeof(uchar4) * totalpixels * CHANNELS);
	// hipMemcpy(*ddepth, *input_image, sizeof(uchar4) * totalpixels * CHANNELS, hipMemcpyHostToDevice);

	// h_3d_points = (double*)malloc(sizeof(double) * totalpixels * 3);
	// unsigned char* depth_image = (unsigned char*)img.data;
	// hipMalloc((void **)&d_depth_image, sizeof(unsigned char) * totalpixels );
	// hipMalloc((void **)&d_3d_points, sizeof(double) * totalpixels * 3 );
	// hipMemcpy(d_depth_image, depth_image, sizeof(unsigned char) * totalpixels , hipMemcpyHostToDevice);
	// unproject_kernel<<<dimGrid,dimBlock>>>(d_depth_image, d_3d_points, rows, cols, cam.cx, cam.cy, cam.fx, cam.fy, cam.fx_inv, cam.fy_inv);
	// hipMemcpy(h_3d_points, d_3d_points, sizeof(double) * totalpixels * 3, hipMemcpyDeviceToHost);
	
	// std::cout<<*(h_3d_points+sizeof(double)+(rows*cols));
	// hipFree(d_depth_image);

      
}

texture<uchar4,hipTextureType2D,hipReadModeNormalizedFloat> tex;



__global__
void test(char *img,int width,int heigth,int channels)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;


    float4 pixel = tex2D(tex, x, y);

	img[(y*width+x)*channels+0] = pixel.x  * 255;
	img[(y*width+x)*channels+1] = pixel.y  * 255;
	img[(y*width+x)*channels+2] = pixel.z  * 255;
	img[(y*width+x)*channels+3] = 0;



}
void rgb_texture_test(cv::Mat img)
{
	cv::resize(img, img, cv::Size(512, 512));

	int rows=img.rows;
	int cols=img.cols;
	int channels=img.channels();
	int width=cols,height=rows,size=rows*cols*channels;

	hipChannelFormatDesc channelDesc=hipCreateChannelDesc<uchar4>();
	hipArray *cuArray;
	hipMallocArray(&cuArray,&channelDesc,width,height);
	hipMemcpyToArray(cuArray,0,0,img.data,size,hipMemcpyHostToDevice);

	tex.addressMode[0]=hipAddressModeWrap; 
	tex.addressMode[1]=hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;  
	tex.normalized =false;          //No normalization

	hipBindTextureToArray(tex,cuArray,channelDesc);


	cv::Mat out=cv::Mat::zeros(width, height, CV_8UC4);
	char *dev_out=NULL;
	hipMalloc((void**)&dev_out, size);

	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

	test <<<dimGrid,dimBlock,0>>>(dev_out,width,height,channels);



    hipMemcpy(out.data,dev_out,size,hipMemcpyDeviceToHost);


    // cv::imwrite("src/MyImage.jpg", out);
    cv::imshow("orignal",img);
    cv::imshow("smooth_image",out);
    cv::waitKey(0);
    printf("saving\n");
    hipFree(dev_out);
    hipFree(cuArray);
    hipUnbindTexture(tex);


}

