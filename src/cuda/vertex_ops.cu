#include "hip/hip_runtime.h"
#include "vertex_ops.cuh"



__global__
void unproject_kernel(unsigned char *depth, double* d_3d_points, int rows, int cols, double cx, double cy, double fx, double fy, double fx_inv, double fy_inv)
{

	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int idy = threadIdx.y + (blockIdx.y * blockDim.y);
	int id;
	if ((idx < rows ) && (idy < cols))
	{
		// add depth info
		id = idx * idy;
		double z = (double)depth[id];
		d_3d_points[id] = (idx-cx)*fx_inv*z;
		d_3d_points[id+(rows*cols)] = (idy-cy)*fy_inv*z;
		d_3d_points[id+2*(rows*cols)] = z;
		// printf("--%f--\n",(cx));
	}
}


void unproject(cv::Mat img, GSLAM::CameraPinhole cam)
{

	// uchar4 **ddepth, **input_image;
	// unsigned char *hdepth;
	unsigned char *d_depth_image;
	double *d_3d_points, *h_3d_points;
	int rows, cols;
	rows = img.rows;
	cols = img.cols;
	size_t totalpixels = rows*cols;
	const dim3 dimGrid((int)ceil((cols)/16), (int)ceil((rows)/16));
	const dim3 dimBlock(16, 16);
	std::cout<<"t";
	// *input_image = (uchar4 *)img.ptr<uchar4 *>(0);
	// hipMalloc(ddepth, sizeof(uchar4) * totalpixels * CHANNELS);
	// hipMemcpy(*ddepth, *input_image, sizeof(uchar4) * totalpixels * CHANNELS, hipMemcpyHostToDevice);

	// h_3d_points = (double*)malloc(sizeof(double) * totalpixels * 3);
	// unsigned char* depth_image = (unsigned char*)img.data;
	// hipMalloc((void **)&d_depth_image, sizeof(unsigned char) * totalpixels );
	// hipMalloc((void **)&d_3d_points, sizeof(double) * totalpixels * 3 );
	// hipMemcpy(d_depth_image, depth_image, sizeof(unsigned char) * totalpixels , hipMemcpyHostToDevice);
	// unproject_kernel<<<dimGrid,dimBlock>>>(d_depth_image, d_3d_points, rows, cols, cam.cx, cam.cy, cam.fx, cam.fy, cam.fx_inv, cam.fy_inv);
	// hipMemcpy(h_3d_points, d_3d_points, sizeof(double) * totalpixels * 3, hipMemcpyDeviceToHost);
	
	// std::cout<<*(h_3d_points+sizeof(double)+(rows*cols));
	// hipFree(d_depth_image);

	texture<double, hipTextureType2D,  hipReadModeElementType> t;
	hipArray* cuArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,
                                             hipChannelFormatKindFloat );
    hipMallocArray(&cuArray, &channelDesc, cols, rows);
    hipMemcpyToArray(cuArray, 0, 0, img.data, sizeof(double)*totalpixels, hipMemcpyHostToDevice);

      
}
__global__
void rgb_texture_kernel(hipTextureObject_t tex_obj, int width, int height)
{
	int x = threadIdx.x + (blockDim.x *blockIdx.x);
	int y = threadIdx.y + (blockDim.y *blockIdx.y);

	float u = x/(float)width;
	float v = y/(float)height;

	u -= 0.5;
	v -= 0.5;

	float t = tex2D<float>(tex_obj, u,v);
	printf("%f\n",t);

}
void rgb_texture_test(cv::Mat img)
{
	int width = img.cols;
	int height = img.rows;
	int size = width * height  * sizeof(float);

	const dim3 dimGrid((int)ceil((width)/16), (int)ceil((height)/16));
	const dim3 dimBlock(16, 16);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
	hipArray* cuArray;
	cudaSafeCall(hipMallocArray(&cuArray, &channelDesc, width, height));
	cudaSafeCall(hipMemcpyToArray(cuArray, 0, 0, img.ptr(), size, hipMemcpyHostToDevice));

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;    
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	rgb_texture_kernel<<<dimGrid, dimBlock>>>(texObj, width, height);

    cudaCheckError();
    hipFreeArray(cuArray);

}