#include "hip/hip_runtime.h"
 // 2D float texture
#include "cuda/cuda_array_test.cuh"

texture<float, hipTextureType2D, hipReadModeElementType> texRef;


// Simple transformation kernel
__global__ void transformKernel(float* output, int width, int height, float theta)
{
      // Calculate normalized texture coordinates
      unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
     
      unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
     
      float u = x / (float)width;
     
      float v = y / (float)height;
     
      // Transform coordinates
      u -= 0.5f;
     
      v -= 0.5f; 
     
      float tu = u * cosf(theta) - v * sinf(theta) + 0.5f ;
     
      float tv = v * cosf(theta) + u * sinf(theta) + 0.5f ;
     
      // Read from texture and write to global memory
      output[y * width + x] = tex2D(texRef, tu, tv);
}


void transform(hipChannelFormatDesc channelDesc,  hipArray* cuArray)
{    


      int width = 3 , height = 3 ;

      float h_data[3][3] ;
      for ( int i =0 ; i<3 ; i++ )
            for ( int j = 0 ; j<3; j++ )
                        h_data [i][j] = i*j ;

      int size = width*height*sizeof(float) ;

     
      hipMallocArray(&cuArray, &channelDesc, width, height);
     
      // Copy to device memory some data located at address h_data
      // in host memory
      hipMemcpyToArray(cuArray, 0, 0, h_data, size, hipMemcpyHostToDevice);
     
      // Set texture parameters
      texRef.addressMode[0] = hipAddressModeWrap;
      texRef.addressMode[1] = hipAddressModeWrap;
      texRef.filterMode = hipFilterModeLinear;
      texRef.normalized = true;
     
      // Bind the array to the texture reference
      hipBindTextureToArray(texRef, cuArray, channelDesc);
     
      // Allocate result of transformation in device memory
      float* output;
     
      hipMalloc(&output, size );
     
      // Invoke kernel
      dim3 dimBlock(16, 16);
      dim3 dimGrid( (width + dimBlock.x - 1) / dimBlock.x,
                       (height + dimBlock.y - 1) / dimBlock.y );
     
      transformKernel<<<dimGrid, dimBlock>>>(output, width, height, 90 );
     
      printf ("\n Original array \n");
      for ( int i =0 ; i<3 ; i++ )
      {
            for ( int j = 0 ; j<3; j++ )
                              printf ("%f ", h_data [i][j] );
           
            printf ("\n");
      }

      hipMemcpy (h_data , output, size, hipMemcpyDeviceToHost );

      printf ("\n After operation \n ");
      for ( int i =0 ; i<3 ; i++ )
      {
            for ( int j = 0 ; j<3; j++ )
                              printf ("%f ", h_data [i][j] );
           
            printf ("\n");
      }

      // Free device memory
      hipFreeArray(cuArray);
      hipFree(output);
}